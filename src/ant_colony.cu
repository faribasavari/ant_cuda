#include "hip/hip_runtime.h"
#include<iostream>
#include<fstream>
#include<math.h>
#include<stdlib.h>
#include<hiprand/hiprand_kernel.h>
#include<hiprand.h>
#define MAX_TIME 2
#define MAX_ANTS 10
#define Q 100
#define ALPHA 2                  //we used alpha 10 beta 1 for u_c_lohi
#define BETA 1
#define RHO 1.5 
#define ntask 512
#define nres 16
#define evaporation 0.5

using namespace std;

// TODO: change the 2D arrays into 1D: solution and makespan
struct ant {
	int curJob, nextJob;
	int visited[ntask];
	int solution[ntask];
	float makespan[nres];
};
struct job {

	double res[nres];
};

int NC = 0;

job jobs[ntask];
double F;
float make[nres];

int allsolution[MAX_TIME][ntask];
float allmakespan[MAX_TIME][nres];
double freeRes[nres];
double pheromone[ntask][nres], Delta[ntask][nres], heuristic[ntask][nres], probability[ntask][nres],valid[ntask][nres];
hiprandState  state[MAX_ANTS];


__global__ void setup_curand_states(hiprandState *state_d, int t) {
	int id = threadIdx.x + blockIdx.x*blockDim.x;
	hiprand_init(t, id, 0, &state_d[id]);
}

__device__ float generate(hiprandState* globalState, int ind) {
	hiprandState localState = globalState[ind];
	float RANDOM = hiprand_uniform(&localState);
	globalState[ind] = localState;
	return RANDOM;
}

__global__ void initialize(float *d_pheromone, float *d_delta, float *d_heuristic, job *d_job, int task, int res, float max)
{
	int col = blockIdx.x * blockDim.x + threadIdx.x;        //res
	int row = blockIdx.y * blockDim.y + threadIdx.y;      //task
	if ((row<task) && (col<res)) {
		d_heuristic[col + row * res] = 1 / max;
		d_delta[col + row * res] = (1 - evaporation) / max;
		d_pheromone[col + row * res] = 0;
		d_pheromone[col + row * res] = evaporation*d_pheromone[col + row * res] + d_delta[col + row * res];
	}

}

__device__ int findmax_probability(float *d_probability, ant *d_ant, job *d_job, float *d_free, int k) {
	int i, j = 0, maxj = 0, maxi = 0;
	double max = d_probability[0];
	for (i = 0; i < ntask; i++) {
		if (d_ant[k].visited[i] == 0)
			for (j = 0; j < nres; j++) {
				if (d_probability[j + nres*i] > max /*&& d_ant[k].tabu[i]==0*/) {
					max = d_probability[j + nres*i];
					maxi = i;
					maxj = j;
				}
			}
	}
	d_ant[k].solution[maxi] = maxj;
	d_ant[k].makespan[maxj] += d_job[maxi].res[maxj];
	for (int g = 0; g < nres; g++) {
		d_probability[g + maxi*nres] = 0;
		d_free[g] = d_free[g] - d_job[maxi].res[g];          //update d_free

	}
	return maxi;
}

__device__ double findmax(float *a) {
	float max = a[0];
	for (int i = 1; i < nres; i++) {
		if (a[i] > max) {
			max = a[i];
		}
	}
	return max;
}

__device__ int selectNextJob(float *d_probability, float *d_pheromone, float *d_delta, float *d_heuristic, job *d_job, int k, int n, float *d_free, ant *d_ant, hiprandState *state_d)
{
	int i;/// = ants[k].curJob;
	int j, nextJob;
	float max, sum = 0;

	max = findmax(d_free);
	for (i = 0; i < ntask; i++) {

		for (j = 0; j < nres; j++) {
			d_heuristic[j + nres*i] = 1 / d_free[j];
			d_delta[j + nres*i] += (1 - evaporation) / max;
			d_pheromone[j + nres*i] = evaporation * d_pheromone[j + nres*i] + d_delta[j + nres*i];
			sum += powf(d_pheromone[j + nres*i], ALPHA) * powf(d_heuristic[j + nres*i], BETA) * 1 / d_job[i].res[j];
		}
	}

	for (i = 0; i < ntask; i++) {            //calculate probability for any task and any resource
		if (d_ant[k].visited[i] == 0) {
			for (j = 0; j < nres; j++) {
				d_probability[j + nres*i] = (powf(d_pheromone[j + nres*i], ALPHA) * powf(d_heuristic[j + nres*i], BETA) * 1 / d_job[i].res[j]) / sum;
			}
		}
	}
	nextJob = findmax_probability(d_probability, d_ant, d_job, d_free, k);

	return nextJob;
}


__global__ void select( float *d_pheromone, float *d_delta, float *d_heuristic, job *d_job, ant *d_ant, float *d_probability, float *d_free, int n, hiprandState *state_d)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < MAX_ANTS) {
		for (int s = 1; s<n; s++)
		{
			int j = selectNextJob(d_probability, d_pheromone, d_delta, d_heuristic, d_job, id, n, d_free, d_ant,state_d);
			//printf("j:%d\n", j);
			d_ant[id].nextJob = j;
			d_ant[id].visited[j] = 1;
			d_ant[id].curJob = j;
		}
	}


}
__global__ void firststep(ant *d_ant, job *d_job) {
	int randres, randtask;
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id<MAX_ANTS) {

		//randtask = (blockIdx.x + clock() + clock() * threadIdx.x *blockDim.x) % ntask;
		// randres=( blockIdx.x + clock() + clock()*threadIdx.x * blockDim.x)%nres;
		hiprandState state;
		hiprand_init((unsigned long long)clock(), id, 0, &state);
		double rand1 = hiprand_uniform_double(&state);
		double rand2 = hiprand_uniform_double(&state);
		randtask = (int)((rand1 / rand2)*blockIdx.x) % ntask;
		randres = (int)((rand1 / rand2)*id) % nres;
		//	printf("rand1:%d , rand2:%d \n", randres, randtask);
		d_ant[id].curJob = randtask;
		for (int i = 0; i<ntask; i++)
		{
			d_ant[id].visited[i] = 0;
		}

		d_ant[id].visited[randtask] = 1;
		for (int i = 0; i<nres; i++)
			d_ant[id].makespan[i] = 0;
		d_ant[id].makespan[randres] = d_job[randtask].res[randres];
		d_ant[id].solution[randtask] = randres;
	}
}

__global__ void emptyTabu(ant *d_ant, float *d_delta, int n) {

	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < n) {
		for (int s = 0; s<n; s++) {

			d_ant[id].visited[s] = 0;
		}
	}
}

__global__ void updatePheromone(float *d_pheromone, float *d_delta, float *d_heuristic, job *d_job, int n, int max) {

	int col = blockIdx.x * blockDim.x + threadIdx.x;        //res
	int row = blockIdx.y * blockDim.y + threadIdx.y;      //task

	if ((row<ntask) && (col<nres)) {
		d_heuristic[col + row * nres] = 1 / max;
		d_delta[col + row * nres] += (1 - evaporation) / max;
		d_pheromone[col + row * nres] = evaporation*d_pheromone[col + row * nres] + d_delta[col + row * nres];
	}
}

int main(int argc, char *argv[])
{
	if (argc > 1) {
		cout << "Reading File " << argv[1] << endl;
	}
	else {
		cout << "Usage:progname inputFileName" << endl;
		return 1;
	}
	int i, j;
	double max;
	ifstream in;

	in.open(argv[1]);
	for (i = 0; i<nres; i++)
	{
		for (j = 0; j<ntask; j++) {

			in >> jobs[j].res[i];
			//cout<<jobs[j].res[i]<<"\t";	
			freeRes[i] = freeRes[i] + jobs[j].res[i];
		}
		cout << endl;
	}
	max = freeRes[0];
	for (i = 1; i < nres; i++) {
		if (freeRes[i] > max) {
			max = freeRes[i];
		}
	}
	clock_t begin = clock();
	dim3 blockDim(16, 16, 1);
	dim3 gridDim((ntask - 1) / 16 + 1, (ntask - 1) / 16 + 1, 1);
	float *d_pheromone, *d_delta, *d_free, *d_heuristic, *d_probability;
	ant *d_ant,ants;
	job *d_job;
	hiprandState  *state_d;
	hipMalloc((void**)&d_pheromone, sizeof(float) * ntask * nres);
	hipMalloc((void**)&d_free, sizeof(float) * nres);
	hipMalloc((void**)&d_delta, sizeof(float) * ntask * nres);
	hipMalloc((void**)&d_ant, sizeof(ant));
	hipMalloc((void**)&d_job, sizeof(job) * ntask);
	hipMalloc((void**)&d_heuristic, sizeof(float) * ntask *nres);
	hipMalloc((void**)&state_d, sizeof(state));
	hipMalloc((void**)&d_probability, sizeof(float) * ntask * nres);
	hipMemcpy(d_job, jobs, sizeof(job) * ntask, hipMemcpyHostToDevice);
	srand(time(0));
	int seed = rand();
	setup_curand_states << < (ntask - 1) / 32 + 1, 32 >> > (state_d, seed);
	initialize <<<gridDim, blockDim >>>(d_pheromone, d_delta, d_heuristic, d_job, ntask, nres, max);
	hipDeviceSynchronize();
	for (;;)
	{	//	cout<<randres<<","<<randtask<<endl;
		firststep <<<(ntask - 1) / 32 + 1, 32 >>>(d_ant, d_job);	

		hipMemcpy(d_free, freeRes, sizeof(float) * nres, hipMemcpyHostToDevice);
		select <<<(ntask - 1) / 32 + 1, 32 >>>(d_pheromone, d_delta, d_heuristic, d_job, d_ant, d_probability, d_free, ntask, state_d);

		hipMemcpy(freeRes, d_free, sizeof(float) * nres, hipMemcpyDeviceToHost);
		max = freeRes[0];
		for (int i = 1; i < nres; i++) {
			if (freeRes[i] > max) {
				max = freeRes[i];
			}
		}
		//hipMemcpy(freeRes, d_free, sizeof(float) * nres, hipMemcpyDeviceToHost);
		updatePheromone <<< (ntask - 1) / 32 + 1, 32 >>>(d_pheromone, d_delta, d_heuristic, d_job, ntask, max);

		
		if (NC < MAX_TIME) {
			emptyTabu <<<(ntask - 1) / 32 + 1, 32 >>>(d_ant, d_delta, ntask);
		//	hipMemcpy(ants, d_ant, sizeof(ant) * MAX_ANTS, hipMemcpyDeviceToHost);
                        hipMemcpy(&ants, d_ant, sizeof(ant), hipMemcpyDeviceToHost); 
                        
	                memcpy(allsolution[NC],ants.solution,sizeof(int)*ntask);
                        memcpy(allmakespan[NC],ants.makespan,sizeof(float)*nres);
                        NC+=1;
		}
		else {
			break;
		}
               
       }//end of for(;;)

	hipDeviceSynchronize();

	//hipMemcpy(ants, d_ant, sizeof(ant) * MAX_ANTS, hipMemcpyDeviceToHost);
	//hipMemcpy(jobs, d_job, sizeof(job) * ntask, hipMemcpyDeviceToHost);

	clock_t end = clock();
	double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;


	/*******************************************************************/

	hipFree(d_free);
	hipFree(d_pheromone);
	hipFree(d_delta);
	hipFree(d_heuristic);
	hipFree(d_probability);
	hipFree(d_job);
	hipFree(d_ant);

	cout<<"_________________\n";
	for (int i = 0; i<MAX_ANTS; i++) {
		max = 0;
		for (int y = 0; y<MAX_TIME; y++) {
			max = allmakespan[y][0];
			//antindex = i;
			//spanindex = y;
			for (int q = 1; q < nres; q++) {
				printf("span:%f\n",allmakespan[y][q]);
				if (allmakespan[y][q] > max) {
					max = allmakespan[y][q];
					//antindex = i;
					//spanindex = y;
				}
			}
			cout << "max makespan" << max << "\t";
			cout << endl;

		}
	}
	cout << "time:" << time_spent;



	return 0;
}
